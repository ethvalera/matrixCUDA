//
//  matrixCUDA.cu
//  
//

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void matrixOperation(int *m, int *v, int *c, int N, int M){
    
    // Calculate global row and column for each thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // AVG column
    if (row < M && col < N) {
        int temp = 0;
        for (int i = 0; i<M; i++){
            temp += m[i * N + col];
        }
        v[col] = temp/M;
    }
    
    // MOV matrix (rolling average)
    if (row < M && col < N) {
        for (int i = 0; i < N; i++){
            int count = 0;
            int tmp = 0;
            for (int j = i; (j>(i-9)) && (j>=0); j--){
                tmp += m[row * N + j];
                count++;
            }
            c[row * N + i] = tmp/count;
            }
        }
}


int main (){
    // Set matrix dimensions
    const int N = 1000;
    const int M = 10;
    const int msize = N * M * sizeof(int);
    const int vsize = N * sizeof(int);
    
    // Allocate memory for matrices
    int *DATA, *AVG, *MOV, *ad, *bd, *cd;
    hipMalloc((void**)&ad, msize);
    hipMalloc((void**)&bd, vsize);
    hipMalloc((void**)&cd, msize);
    hipMalloc((void**)&DATA, msize);
    hipMalloc((void**)&AVG, vsize);
    hipMalloc((void**)&MOV, msize);
    
    // Init. DATA randomly
    for(int i = 0; i < M; i++) {
        for(int j = 0; j < N; j++) {
            DATA[N*i+j] = rand() % 100;
        }
    }

    // Copy memory to GPU
    hipMemcpy(ad, DATA, msize, hipMemcpyHostToDevice);
    hipMemcpy(bd, AVG, vsize, hipMemcpyHostToDevice);
    hipMemcpy(cd, MOV, msize, hipMemcpyHostToDevice);
    
    // Number of threads per block
    int THREADS_M = 16;
    
    // Number of blocks for matrix
    int blocks_rows = (M + THREADS_M - 1) / THREADS_M;
    int blocks_col = (N + THREADS_M - 1) / THREADS_M;
    
    // 2D blocks size for matrix
    dim3 dimBlock(THREADS_M, THREADS_M);
    
    // 2D grid for matrix
    dim3 dimGrid(blocks_col, blocks_rows);
    
    // Call the matrixOperation kernel
    matrixOperation<<<dimGrid, dimBlock>>>(ad, bd, cd, N, M);
    
    // Copy memory back to the CPU
    hipMemcpy(AVG, bd, vsize, hipMemcpyDeviceToHost);
    hipMemcpy(MOV, cd, msize, hipMemcpyDeviceToHost);
    hipFree(bd);
    hipFree(cd);
    
    return EXIT_SUCCESS;
}

